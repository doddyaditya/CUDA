
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <time.h>
#include <assert.h>

__device__ int minDistance(int dist[], int sptSet[], int V)
{
	// Initialize min value
	int min = INT_MAX, min_index;
	for (int v = 0; v < V; v++)
		if (sptSet[v] == 0 && dist[v] <= min)
			min = dist[v], min_index = v;

	return min_index;
}
__global__ void dijkstra(int *graph, int V,int* ansArray)
{
	int nodes = blockDim.x * blockIdx.x + threadIdx.x;
	if(nodes<V)
	{	int dist[3000]; // The output array. dist[i] will hold the shortest
		// distance from src to i

		int sptSet[3000]; // sptSet[i] will be true if vertex i is included in shortest
		// path tree or shortest distance from src to i is finalized

		// Initialize all distances as INFINITE and stpSet[] as false
		for (int i = 0; i < V; i++)
			dist[i] = INT_MAX, sptSet[i] = 0;

		// Distance of source vertex from itself is always 0
		dist[nodes] = 0;

		// Find shortest path for all vertices
		for (int count = 0; count < V - 1; count++)
		{
			// Pick the minimum distance vertex from the set of vertices not
			// yet processed. u is always equal to src in the first iteration.

			int u = minDistance(dist, sptSet, V);

			// Mark the picked vertex as processed
			sptSet[u] = 1;

			// Update dist value of the adjacent vertices of the picked vertex.
			for (int v = 0; v < V; v++)

				// Update dist[v] only if is not in sptSet, there is an edge from
				// u to v, and total weight of path from src to v through u is
				// smaller than current value of dist[v]
				if (!sptSet[v] && graph[u*V+v] && dist[u] != INT_MAX && dist[u] + graph[u*V+v] < dist[v])
					dist[v] = dist[u] + graph[u*V+v];
		}
		for (int i = 0; i < V; i++)
		{
			ansArray[nodes*V+i] = dist[i];
		}
	}
}
__host__ int* initGraf(int n)
{
	srand(13517143);
	int random;
	int *graf=(int *)malloc(n*n* sizeof(int ));
	for (int i = 0; i < n; i++)
	{
		for (int j = i; j < n; j++)
		{
			random = rand() % 100;
			if (i == j)
			{
				graf[i*n + j] = 0;
			}
			else
			{
				graf[i*n + j] = random;
				graf[j*n + i] = random;
			}
		}
	}
	return graf;
}

int main(int argc, char *argv[])
{
	int thread_count = strtol(argv[1], NULL, 10);
	int node_count = strtol(argv[2],NULL,10);
	int *graf,*answerMatrix,*deviceGraf,*deviceResult;
	graf= initGraf(node_count);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	answerMatrix= (int *)malloc(node_count *node_count* sizeof(int));
	hipMalloc(&deviceGraf,node_count*node_count*sizeof(int));
	hipMalloc(&deviceResult,node_count*node_count*sizeof(int));
	hipMemcpy(deviceGraf, graf, node_count*node_count*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(start);

	dijkstra<<<(node_count/thread_count)+1,thread_count>>>(deviceGraf,node_count,deviceResult);
	hipEventRecord(stop);


	hipMemcpy(answerMatrix, deviceResult, node_count*node_count*sizeof(int), hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time elapsed = %f microseconds\n",milliseconds*100);
	printf("Answer Graf\n");

	for (int i = 0; i < node_count; i++) {
		for (int j = 0; j < node_count; j++) 
		  printf("%d\t", answerMatrix[i*node_count + j]);
		printf("\n");
	  }
	
	hipFree(deviceResult);
	hipFree(deviceGraf);


	
}
